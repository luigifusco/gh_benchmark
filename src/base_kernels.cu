#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime.h>

namespace cg = cooperative_groups;

/**
 * these functions are meant to evaluate typical memory access pattern from device
 * memory should be prepared in advance in such a way that makes these kernels meaningful
 */

template <typename T, unsigned int STRIDE>
__global__ void strided_write_kernel(T *out) {
    auto tid = cg::this_grid().thread_rank();

    out[tid * STRIDE] = 0;
}

template <typename T, unsigned int STRIDE>
__global__ void strided_read_kernel(const T *in) {
    auto tid = cg::this_grid().thread_rank();
    T local;
    const T *target_address = in + tid;

    assert(!*target_address);
}

template <typename T, unsigned int STRIDE>
__global__ void strided_copy_kernel(T *out, const T *in) {
    auto tid = cg::this_grid().thread_rank();

    out[tid * STRIDE] = in[tid * STRIDE];
}

template <typename T, unsigned int STRIDE>
__global__ void strided_sum_kernel(T *out, const T *a, const T *b) {
    auto tid = cg::this_grid().thread_rank();

    out[tid * STRIDE] = a[tid * STRIDE] + b[tid * STRIDE];
}

__global__ void pointer_chase_kernel(unsigned long long int *ptr) {
    while (ptr) {
        ptr = (unsigned long long int *) *ptr;
    }
}

__global__ void atomic_cas_pointer_chase_kernel(unsigned long long int *ptr) {
    while (ptr) {
        ptr = (unsigned long long int *) atomicCAS(ptr, 0, 0);
    }
}

// LAUNCH WITH ONLY ONE BLOCK!
__global__ void ping_pong_receive_first_kernel(void *to_send, void *received, void *send_buffer, bool *send_canary, void *recv_buffer, bool *recv_canary, size_t buffer_size) {
    auto tid = cg::this_thread_block().thread_rank();

    if (tid == 0) {
        while (!*recv_canary) {}
    }

    __syncthreads();

    for (size_t i = tid; i < buffer_size / sizeof(uint64_t); i += cg::this_thread_block().size()) {
        ((uint64_t *) received)[i] = ((uint64_t *) recv_buffer)[i];
    }

    for (size_t i = tid; i < buffer_size / sizeof(uint64_t); i += cg::this_thread_block().size()) {
        ((uint64_t *) send_buffer)[i] = ((uint64_t *) to_send)[i];
    }

    cuda::atomic_thread_fence(cuda::memory_order_seq_cst, cuda::thread_scope_system);

    __syncthreads();

    if (tid == 0) {
        *send_canary = true;
    }
}
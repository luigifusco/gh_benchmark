
#include <hip/hip_runtime.h>
#include <stdio.h>

__attribute__((always_inline)) __device__ inline clock_t get_gpu_clock() {
    unsigned long long tsc;

    asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(tsc));

    return tsc;
}

__global__ void wait_kernel(clock_t *t) {
    clock_t end = get_gpu_clock() + 1000000000;
    while (get_gpu_clock() < end) {}
    *t = get_gpu_clock();
}

int main() {
    hipEvent_t start, end;
    clock_t t;

    hipEventCreate(&start);
    hipEventCreate(&end);

    hipEventRecord(start);
    wait_kernel<<<1, 1>>>(&t);
    hipEventRecord(end);

    hipEventSynchronize(end);

    float ms;

    hipEventElapsedTime(&ms, start, end);

    printf("%f\n", ms);

    return 0;
}